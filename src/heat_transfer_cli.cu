#include "init.hpp"
#include "io.hpp"
#include "update.hpp"
#include "util.hpp"
#include <iostream>

#define CONFIGURATIONS_STRING "1=straighforward unified, 2=straighforward standard"

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

void serial(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next);
void straightforward_unified(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next, dim3 block_dim);
void straightforward_standard(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next, dim3 block_dim);
void tiled_no_halos(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next, dim3 block_dim);
void tiled_with_halos(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next, dim3 block_dim);

void print_usage(const char* program_name);

int main(int argc, char *argv[])
{

    unsigned int configuration;
    unsigned int block_dim_x;
    unsigned int block_dim_y;

    --argc;

    if(argc == 0)
    {
        configuration = 0;
    }
    else if(argc == 1)
    {
        if(strncmp(argv[1], "-h", 2) == 0 || strncmp(argv[1], "--help", 6) == 0)
        {
            print_usage(argv[0]);
            return 0;
        }
    }
    else if(argc == 3 && strncmp(argv[3], "--conf=", 7) == 0)
    {
        block_dim_x = std::stoi(argv[1]);
        block_dim_y = std::stoi(argv[2]);

        if(block_dim_x == 0 || block_dim_y == 0)
        {
            std::cerr << "Invalid block dimensions" << std::endl;
            print_usage(argv[0]);
            return 1;
        }

        if(strncmp(argv[3], "--conf=", 7) != 0)
        {
            std::cerr << "Invalid configuration" << std::endl;
            print_usage(argv[0]);
            return 1;
        }

        configuration = std::stoi(argv[3] + 7);

        if(configuration == 0)
        {
            std::cerr << "Invalid configuration" << std::endl;
            print_usage(argv[0]);
            return 1;
        }
    }

    unsigned int step = 0;
    unsigned int n_steps {10000};
    unsigned int grid_rows {1 << 8};
    unsigned int grid_cols {1 << 12};
    unsigned int n_hot_top_rows {2};
    unsigned int n_hot_bottom_rows {2};
    double initial_hot_temperature {20};
    double * temperature_current = new (std::nothrow) double[grid_rows*grid_cols];
    double * temperature_next = new (std::nothrow) double[grid_rows*grid_cols];
    double elapsed_time {0.0};
    unsigned int field_width {5};
    std::string outfile_prefix {"temperature"};
    std::string outfile_extension {".dat"};
 
    init_top_bottom_temperature(temperature_current, grid_rows, grid_cols, n_hot_top_rows, n_hot_bottom_rows, initial_hot_temperature);
    init_top_bottom_temperature(temperature_next, grid_rows, grid_cols, n_hot_top_rows, n_hot_bottom_rows, initial_hot_temperature);

    std::cout << "Grid size: " << grid_rows << " x " << grid_cols << std::endl;
    std::cout << "Number of steps: " << n_steps << std::endl;

    if (configuration != 0)
    {
        std::cout << "Block dimensions: " << block_dim_x << " x " << block_dim_y << std::endl;
    }

    std::cout << "Configuration: " << configuration << std::endl;

    std::cout << "Saving initial configuration... " << std::endl;
    save_temperature(outfile_prefix, outfile_extension, step, temperature_current, grid_rows, grid_cols, field_width);
    std::cout << "Done" << std::endl;

    std::cout << "Simulation in progress... " << std::endl;
    util::Timer clTimer;

    // Check for CUDA devices
    int device_count;
    cudaCheckError(hipGetDeviceCount(&device_count));

    dim3 block_dim(block_dim_x, block_dim_y);

    // Execute selected configuration
    switch (configuration) {
        case 0:
            serial(n_steps, grid_rows, grid_cols, n_hot_top_rows, n_hot_bottom_rows, temperature_current, temperature_next);
            break;
        case 1:
            straightforward_unified(n_steps, grid_rows, grid_cols, n_hot_top_rows, n_hot_bottom_rows, temperature_current, temperature_next, block_dim);
            break;
        case 2:
            straightforward_standard(n_steps, grid_rows, grid_cols, n_hot_top_rows, n_hot_bottom_rows, temperature_current, temperature_next, block_dim);
            break;
        case 3:
            tiled_no_halos(n_steps, grid_rows, grid_cols, n_hot_top_rows, n_hot_bottom_rows, temperature_current, temperature_next, block_dim);
            break;
        case 4:
            tiled_with_halos(n_steps, grid_rows, grid_cols, n_hot_top_rows, n_hot_bottom_rows, temperature_current, temperature_next, block_dim);
            break;
    }

    elapsed_time = static_cast<double>(clTimer.getTimeMilliseconds());
    std::cout << "Simulation loop elapsed time: " << elapsed_time << " ms (corresponding to " << (elapsed_time / 1000.0) << " s)" << std::endl;

    std::cout << "Saving final configuration... " << std::endl;
    save_temperature(outfile_prefix, outfile_extension, --step, temperature_current, grid_rows, grid_cols, field_width); 
    std::cout << "Done" << std::endl;

    /*
     * To visualize the simulation outcome, run gnuplot and use the following command:
     *
     *   plot 'temperature_step_N.dat' matrix with image
     *
     * where N is the step of the final configuration. Use quit to exit gnuplot.
     *
     */

    delete[] temperature_current, temperature_next;
    return 0;
}

void serial(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next)
{
    unsigned int step;
    for (step = 1; step <= n_steps; step++)
    {
        update_region(temperature_next, temperature_current, grid_rows, grid_cols, n_hot_top_rows, (grid_rows-1)-n_hot_bottom_rows, 1, (grid_cols-1)-1);
        swap_buffer_ptrs(temperature_next, temperature_current);    
    }
}

void straightforward_unified(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next, dim3 block_dim)
{
    /** 
        This function uses automatic unified memory management, which allows the CPU and GPU to access the same memory
        without explicit memory transfers. The CPU and GPU can access the same memory location, which is allocated using
        hipMallocManaged.
    */

    double *d_temp_current, *d_temp_next;

    hipMallocManaged(&d_temp_current, grid_rows * grid_cols * sizeof(double));
    hipMallocManaged(&d_temp_next, grid_rows * grid_cols * sizeof(double));

    memcpy(d_temp_current, temperature_current, grid_rows * grid_cols * sizeof(double));
    memcpy(d_temp_next, temperature_next, grid_rows * grid_cols * sizeof(double));

    dim3 grid_dim(
        (grid_cols + block_dim.x - 1) / block_dim.x,
        (grid_rows + block_dim.y - 1) / block_dim.y
    );

    for (unsigned int step = 1; step <= n_steps; step++)
    {
        straightforward_unified_kernel<<<grid_dim, block_dim>>>( d_temp_next, d_temp_current, grid_rows, grid_cols, n_hot_top_rows, (grid_rows-1)-n_hot_bottom_rows, 1, (grid_cols-1)-1);
        
        // Check for any errors launching the kernel
        cudaCheckError(hipGetLastError());

        // Synchronize the device and check for synchronization errors
        cudaCheckError(hipDeviceSynchronize());

        swap_buffer_ptrs(d_temp_next, d_temp_current);
    }

    memcpy(temperature_current, d_temp_current, grid_rows * grid_cols * sizeof(double));
    memcpy(temperature_next, d_temp_next, grid_rows * grid_cols * sizeof(double));

    cudaCheckError(hipFree(d_temp_current));
    cudaCheckError(hipFree(d_temp_next));
}

void straightforward_standard(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next, dim3 block_dim)
{
    /**
        This function uses standard device / host memory management.
    */

    double *d_temp_current, *d_temp_next;

    cudaCheckError(hipMalloc(&d_temp_current, grid_rows * grid_cols * sizeof(double)));
    cudaCheckError(hipMalloc(&d_temp_next, grid_rows * grid_cols * sizeof(double)));

    cudaCheckError(hipMemcpy(d_temp_current, temperature_current, grid_rows * grid_cols * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_temp_next, temperature_next, grid_rows * grid_cols * sizeof(double),  hipMemcpyHostToDevice));

    dim3 grid_dim(
        (grid_cols + block_dim.x - 1) / block_dim.x,
        (grid_rows + block_dim.y - 1) / block_dim.y
    );

    for (unsigned int step = 1; step <= n_steps; step++)
    {
        straightforward_unified_kernel<<<grid_dim, block_dim>>>(d_temp_next, d_temp_current, grid_rows, grid_cols, n_hot_top_rows, (grid_rows-1)-n_hot_bottom_rows, 1, (grid_cols-1)-1);
        
        cudaCheckError(hipGetLastError());
        cudaCheckError(hipDeviceSynchronize());

        swap_buffer_ptrs(d_temp_next, d_temp_current);
    }

    // Copy final results back to host
    cudaCheckError(hipMemcpy(temperature_current, d_temp_current, grid_rows * grid_cols * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(temperature_next, d_temp_next, grid_rows * grid_cols * sizeof(double), hipMemcpyDeviceToHost));

    cudaCheckError(hipFree(d_temp_current));
    cudaCheckError(hipFree(d_temp_next));
}

void tiled_no_halos(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next, dim3 block_dim)
{
    /**
        This function uses standard device / host memory management. The kernel uses shared memory to store the tile so that
        the global memory accesses are reduced.
    */

    if(block_dim.x < 16 || block_dim.y < 16)
    {
        std::cerr << "Block dimensions must be at least 16 x 16" << std::endl;
        return;
    }

    double *d_temp_current, *d_temp_next;

    cudaCheckError(hipMalloc(&d_temp_current, grid_rows * grid_cols * sizeof(double)));
    cudaCheckError(hipMalloc(&d_temp_next, grid_rows * grid_cols * sizeof(double)));

    cudaCheckError(hipMemcpy(d_temp_current, temperature_current, grid_rows * grid_cols * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_temp_next, temperature_next, grid_rows * grid_cols * sizeof(double),  hipMemcpyHostToDevice));

    dim3 grid_dim(
        (grid_cols + block_dim.x - 1) / block_dim.x,
        (grid_rows + block_dim.y - 1) / block_dim.y
    );

    for (unsigned int step = 1; step <= n_steps; step++)
    {
        tiled_no_halos_kernel<<<grid_dim, block_dim>>>(d_temp_next, d_temp_current, grid_rows, grid_cols, n_hot_top_rows, (grid_rows-1)-n_hot_bottom_rows, 1, (grid_cols-1)-1);
        
        cudaCheckError(hipGetLastError());
        cudaCheckError(hipDeviceSynchronize());

        swap_buffer_ptrs(d_temp_next, d_temp_current);
    }

    // Copy final results back to host
    cudaCheckError(hipMemcpy(temperature_current, d_temp_current, grid_rows * grid_cols * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(temperature_next, d_temp_next, grid_rows * grid_cols * sizeof(double), hipMemcpyDeviceToHost));

    cudaCheckError(hipFree(d_temp_current));
    cudaCheckError(hipFree(d_temp_next));
}

void tiled_with_halos(unsigned int n_steps, unsigned int grid_rows, unsigned int grid_cols, unsigned int n_hot_top_rows, unsigned int n_hot_bottom_rows, double* temperature_current, double* temperature_next, dim3 block_dim)
{
    /**
        This function uses standard device / host memory management. The kernel uses shared memory to store the tile from neighboring
        and also the halo elements so that the global memory accesses are reduced. 
    */

    if(block_dim.x < 16 || block_dim.y < 16)
    {
        std::cerr << "Block dimensions must be at least 16 x 16" << std::endl;
        return;
    }

    double *d_temp_current, *d_temp_next;

    cudaCheckError(hipMalloc(&d_temp_current, grid_rows * grid_cols * sizeof(double)));
    cudaCheckError(hipMalloc(&d_temp_next, grid_rows * grid_cols * sizeof(double)));

    cudaCheckError(hipMemcpy(d_temp_current, temperature_current, grid_rows * grid_cols * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_temp_next, temperature_next, grid_rows * grid_cols * sizeof(double),  hipMemcpyHostToDevice));

    dim3 grid_dim(
        (grid_cols + block_dim.x - 1) / block_dim.x,
        (grid_rows + block_dim.y - 1) / block_dim.y
    );

    for (unsigned int step = 1; step <= n_steps; step++)
    {
        tiled_with_halos_kernel<<<grid_dim, block_dim>>>(d_temp_next, d_temp_current, grid_rows, grid_cols, n_hot_top_rows, (grid_rows-1)-n_hot_bottom_rows, 1, (grid_cols-1)-1);
        
        cudaCheckError(hipGetLastError());
        cudaCheckError(hipDeviceSynchronize());

        swap_buffer_ptrs(d_temp_next, d_temp_current);
    }

    // Copy final results back to host
    cudaCheckError(hipMemcpy(temperature_current, d_temp_current, grid_rows * grid_cols * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(temperature_next, d_temp_next, grid_rows * grid_cols * sizeof(double), hipMemcpyDeviceToHost));

    cudaCheckError(hipFree(d_temp_current));
    cudaCheckError(hipFree(d_temp_next));
}

void print_usage(const char* program_name)
{
    std::cout << "Usage: " << program_name << " [block_dim_x] [block_dim_y] [--conf=N]\n"
            << "Options:\n"
            << "  block_dim_x     Block dimension x\n"
            << "  block_dim_y     Block dimension y\n"
            << "  --conf=N        Configuration to use ("<< CONFIGURATIONS_STRING <<")\n"
            << "  -h, --help      Display this help message\n"
            << "To run the program with serial (0) configuration, do not provide any arguments\n";
}